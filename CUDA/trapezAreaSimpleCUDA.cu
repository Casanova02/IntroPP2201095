/*
 * Algoritmo para solucionar integrales por la regla del trapecio. Esta algoritmo se ejecuta en 
 * paralelo, usando la plataforma CUDA.
 *
 * Autor: ThesplumCoder.
 * Hecho: 23/06/2023.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
 
/* Define la funcion que va a ser integrada aqui */
double f(double x){
  return x*x;
}

/*
 * subInt: numero de sub-intervalos.
 * limIni: limite inicial.
 * tamIntr: tamaño del intervalo.
 * sum: suma acumulada.
 */
__global__ void sumReglaTrapecio (int* subInt, double* limIni, double* tamIntr, double* sum) {
    // idx: Indice del hilo
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int x;
    if (idx >= 1 && idx <= (*subInt - 1)) {
       x = *limIni + idx * (*tamIntr);
       *sum += (x * x);
    }
    __syncthreads();
}
 
/* Algoritmo empieza */
int main(int argc, char* argv[]) {
    /* n: numero de subintervalos.
     * i:
     * a: limite inicial.
     * b: limite final.
     * h: tamahno del intervalo.
     */
    hipError_t errorDevice = hipSuccess;
    int n,i;
    double a,b,h,x,sum=0,integral;

    // Captura de datos pasados por consola.
    n = (int) atoi(argv[1]);
    a = (double) atof(argv[2]);
    b = (double) atof(argv[3]);
    h = (b - a) / (double)n;

    printf("Enter the no. of sub-intervals: %i\n", n);
    printf("Enter the initial limit: %lf\n", a);
    printf("Enter the final limit: %lf\n", b);

    // Hacemos punteros para cada variable del host.
    int* host_n = &n;
    double* host_a = &a;
    double* host_h = &h;
    double* host_sum = &sum;
    if (host_n == NULL || host_a == NULL|| host_h == NULL|| host_sum == NULL) {
        printf("Ocurrio un error con los punteros del host");
        exit(-1);
    }

    // Reservamos memoria para cada variable en el device.
    int* device_n = NULL;
    errorDevice = hipMalloc((void**)&device_n, sizeof(int));
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error reservando memoria del device");
        exit(-1);
    }
    double* device_a = NULL;
    errorDevice = hipMalloc((void**)&device_a, sizeof(double));
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error reservando memoria del device");
        exit(-1);
    }
    double* device_h = NULL;
    errorDevice = hipMalloc((void**)&device_h, sizeof(double));
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error reservando memoria del device");
        exit(-1);
    }
    double* device_sum = NULL;
    errorDevice = hipMalloc((void**)&device_sum, sizeof(double));
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error reservando memoria del device");
        exit(-1);
    }

    // Copiamos los datos del host al device.
    errorDevice = hipMemcpy(device_n, host_n, sizeof(int), hipMemcpyHostToDevice);
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error copiando del host al device");
        exit(-1);
    }
    errorDevice = hipMemcpy(device_a, host_a, sizeof(double), hipMemcpyHostToDevice);
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error copiando del host al device");
        exit(-1);
    }
    errorDevice = hipMemcpy(device_h, host_h, sizeof(double), hipMemcpyHostToDevice);
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error copiando del host al device");
        exit(-1);
    }
    errorDevice = hipMemcpy(device_sum, host_sum, sizeof(double), hipMemcpyHostToDevice);
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error copiando del host al device");
        exit(-1);
    }

    // Ejecutamos el kernel.
    int hilosPorBloque = 256;
    int bloquesPorRed = 1;
    sumReglaTrapecio<<<bloquesPorRed, hilosPorBloque>>>(device_n, device_a, device_h, device_sum);
    errorDevice = hipGetLastError();

    // Nos tremos el resultado de la suma.
    errorDevice = hipMemcpy(host_sum, device_sum, sizeof(double), hipMemcpyDeviceToHost);
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error copiando del device al host\n");
        //printf("%s\n", cudaGetErrorString(errorDevice));
        printf("%s\n", errorDevice);
        exit(-1);
    }

    // Liberamos la memoria en el device.
    errorDevice = hipFree(device_n);
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error liberando memoria del device\n");
        printf("%s\n", errorDevice);
        exit(-1);
    }
    errorDevice = hipFree(device_a);
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error liberando memoria del device\n");
        printf("%s\n", errorDevice);
        exit(-1);
    }
    errorDevice = hipFree(device_h);
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error liberando memoria del device\n");
        printf("%s\n", errorDevice);
        exit(-1);
    }
    errorDevice = hipFree(device_sum);
    if (errorDevice != hipSuccess) {
        printf("Ocurrio un error liberando memoria del device\n");
        printf("%s\n", errorDevice);
        exit(-1);
    }
    
    // Reseteamos el device.
    errorDevice = hipDeviceReset();

    // Imprimimos la suma.
    //printf("Suma del kernel: %lf", *host_sum);
    integral = h * (((f(a) - f(b)) / 2) + *host_sum);
    printf("Integral: %lf", integral);

    // Liberamos la memoria del host.
    /*
    free(host_n);
    free(host_a);
    free(host_h);
    free(host_sum);
    */
}

