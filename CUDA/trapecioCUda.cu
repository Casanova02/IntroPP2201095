
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// Define the function to be integrated here:
__host__ __device__ double f(double x){
  return x*x;
}

__device__ void atomicAddDouble(double* address, double val){
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
}

__global__ void trapezoidalRule(int n, double a, double b, double h, double* integral){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double x, sum = 0.0;

  for(int i = tid + 1; i < n; i += blockDim.x * gridDim.x){
    x = a + i * h;
    sum += f(x);
  }

  atomicAddDouble(integral, sum);
}

int main(){
  int n;
  double a, b, h, integral = 0.0;
  double *d_integral;

  // Ask the user for necessary input
  printf("\nEnter the no. of sub-intervals: ");
  scanf("%d", &n);
  printf("\nEnter the initial limit: ");
  scanf("%lf", &a);
  printf("\nEnter the final limit: ");
  scanf("%lf", &b);

  // Calculate step size
  h = fabs(b - a) / n;

  // Allocate memory on the device for the integral
  hipMalloc((void**)&d_integral, sizeof(double));
  hipMemcpy(d_integral, &integral, sizeof(double), hipMemcpyHostToDevice);

  // Set grid and block dimensions
  int blockSize = 256;
  int gridSize = (n + blockSize - 1) / blockSize;

  // Launch kernel to perform trapezoidal rule
  trapezoidalRule<<<gridSize, blockSize>>>(n, a, b, h, d_integral);

  // Copy the result back to the host
  hipMemcpy(&integral, d_integral, sizeof(double), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_integral);

  // Multiply by h/2 and add the first and last terms
  integral = (h/2) * (f(a) + f(b) + 2 * integral);

  // Print the answer
  printf("\nThe integral is: %lf\n", integral);
}


