
#include <hip/hip_runtime.h>
#include <stdio.h>

//Aqui definimos nuestro kernel, donde el parametro va a ser la referencia de la direccion de la variable dev_sum y el numero que vamos a sumar
//tid representa el identificador unico de cada hilo

/*La expresión threadIdx.x devuelve el índice del hilo dentro de su bloque, mientras que blockIdx.x devuelve el índice del bloque dentro de la cuadrícula.

La multiplicación blockIdx.x * blockDim.x se utiliza para calcular el desplazamiento necesario para asignar un identificador único a cada hilo en la cuadrícula completa.

Sumando threadIdx.x y blockIdx.x * blockDim.x, obtenemos un valor único para cada hilo en la cuadrícula */
__global__ void sumParallel(int *dev_sum, int num)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

	/*stride se puede definir como el paso y se calcula se calcula como el producto entre blockDim.x y gridDim.x Esto permite distribuir uniformemente la
	 carga de trabajo entre los hilos en todos los bloques de la cuadrícula */
    int stride = blockDim.x * gridDim.x;

    while (tid <= num)
    {

	/* se usa la función atomicAdd para realizar la sumatoria y  garantizar que varios hilos no vayan a escribir simultáneamente en la misma ubicación de memoria. */
        atomicAdd(dev_sum, tid);
        tid += stride; //coge el identificado unico de cada hilo y le agrega el paso para pasar al siguiente hilo
    }
}

int main()
{
    int num, sum = 0;
    int *dev_sum; //referencia de la variable que se ubicara en el device

    /*Inicializamos las variables con las cuales tomaremos el tiempo */
    hipEvent_t start, stop;
    float elapsedTime;


    printf("Enter a positive integer: ");
    scanf("%d", &num);


    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0); //comienza a tomar el tiempo
    
    hipMalloc((void**)&dev_sum, sizeof(int)); //reservamos espacio de memoria
    hipMemcpy(dev_sum, &sum, sizeof(int), hipMemcpyHostToDevice); //copaimos la variable desde el host al sum

    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize;

    sumParallel<<<gridSize, blockSize>>>(dev_sum, num); //invocamos el kernel sumParallel que es el que se encarga de realizar la suma

    hipMemcpy(&sum, dev_sum, sizeof(int), hipMemcpyDeviceToHost); //copiamos el resultado ahora en sentido contrario, es decir desde el device hasta el host
    hipFree(dev_sum); //liberamos la memoria reservada

    printf("\nSum = %d\n", sum); //imprimimos el resultado de la suma
    
    hipEventRecord(stop, 0); //para de tomar el tiempo
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Elapsed Time: %.6f segundos\n", elapsedTime/1000); //me imprime el tiempo que demoro

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
