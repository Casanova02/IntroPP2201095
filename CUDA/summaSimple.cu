
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sumParallel(int *dev_sum, int num)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (tid <= num)
    {
        atomicAdd(dev_sum, tid);
        tid += stride;
    }
}

int main()
{
    int num, sum = 0;
    int *dev_sum;

    printf("Enter a positive integer: ");
    scanf("%d", &num);

    hipMalloc((void**)&dev_sum, sizeof(int));
    hipMemcpy(dev_sum, &sum, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize;

    sumParallel<<<gridSize, blockSize>>>(dev_sum, num);

    hipMemcpy(&sum, dev_sum, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_sum);

    printf("\nSum = %d\n", sum);

    return 0;
}
