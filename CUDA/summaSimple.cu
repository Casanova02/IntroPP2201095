#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void sumParallel(int *dev_sum, int num)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int localSum = 0;

    while (tid <= num)
    {
        localSum += tid;
        tid += stride;
    }

    atomicAdd(dev_sum, localSum);
}

int main()
{
    int num, sum = 0;
    int *dev_sum;

    printf("Enter a positive integer: ");
    scanf("%d", &num);

    hipMalloc((void**)&dev_sum, sizeof(int));
    hipMemcpy(dev_sum, &sum, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize;

    sumParallel<<<gridSize, blockSize>>>(dev_sum, num);

    hipMemcpy(&sum, dev_sum, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_sum);

    printf("\nSum = %d\n", sum);

    return 0;
}
